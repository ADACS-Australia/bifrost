/*
 * Copyright (c) 2016, The Bifrost Authors. All rights reserved.
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * * Redistributions of source code must retain the above copyright
 *   notice, this list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimer in the
 *   documentation and/or other materials provided with the distribution.
 * * Neither the name of The Bifrost Authors nor the names of its
 *   contributors may be used to endorse or promote products derived
 *   from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*! \file testfft.cu
 *  \brief This file tests the fft.cu functionality.
 */

#include "fft.cu"
#include <assert.hpp>

BFstatus test_bffft_real_2d()
{
    BFarray my_data;
    BFarray out_data;
    BFreal set_data[3][2] = 
        {{1,2},{2,3},{3,4}};
    BFreal** some_data;
    BFcomplex* odata;
    hipMalloc((void**)&some_data, sizeof(BFreal)*6);
    hipMalloc((void**)&odata, sizeof(BFcomplex)*6);
    hipMemcpy(
        some_data, set_data, 
        sizeof(BFreal)*6, hipMemcpyHostToDevice);
    my_data.data = some_data;
    my_data.space = BF_SPACE_CUDA;
    my_data.shape[0] = 3;
    my_data.shape[1] = 2;
    my_data.dtype = 0;
    my_data.ndim = 2;
    my_data.strides[0] = 2*sizeof(BFreal);
    my_data.strides[1] = sizeof(BFreal);
    out_data = my_data;
    out_data.data = odata;
    out_data.dtype = 1;
    out_data.strides[0] = 2*sizeof(BFcomplex);
    out_data.strides[1] = sizeof(BFcomplex);
    if (bfFFT(&my_data, &out_data, FFT_FORWARD) != BF_STATUS_SUCCESS)
    {
        return BF_STATUS_INTERNAL_ERROR; 
    }
    hipfftComplex localdata[3][2] = {};
    hipMemcpy(
        localdata, (hipfftComplex*)out_data.data, 
        sizeof(hipfftComplex)*6, hipMemcpyDeviceToHost);
    BF_ASSERT(hipCrealf(localdata[0][0])==15.2,0);
    printf("Still good?");
    return BF_STATUS_SUCCESS;
}
void test_bffft_real()
{
    BFarray my_data;
    BFarray out_data;
    BFreal set_data[4] = {1,3,6,2.5134};
    BFreal* some_data;
    BFcomplex* odata;
    hipMalloc((void**)&some_data, sizeof(BFreal)*5);
    hipMalloc((void**)&odata, sizeof(BFcomplex)*3);
    hipMemcpy(
        some_data, set_data, 
        sizeof(BFreal)*4, hipMemcpyHostToDevice);
    my_data.data = some_data;
    my_data.space = BF_SPACE_CUDA;
    my_data.shape[0] = 4;
    my_data.dtype = 0;
    my_data.ndim = 1;
    my_data.strides[0] = sizeof(BFreal);
    out_data = my_data;
    out_data.data = odata;
    out_data.dtype = 1;
    out_data.strides[0] = sizeof(BFcomplex);
    if (bfFFT(&my_data, &out_data, FFT_FORWARD) != BF_STATUS_SUCCESS)
    {
        printf("bfFFT failed!\n");
        return; 
    }
    hipfftComplex localdata[3] = {};
    hipMemcpy(
        localdata, (hipfftComplex*)out_data.data, 
        sizeof(hipfftComplex)*3, hipMemcpyDeviceToHost);
    for(int i = 0; i < 3; i++)
        printf("%f+I%f\n",hipCrealf(localdata[i]),hipCimagf(localdata[i]));
    return;
}

void test_bffft_2d()
{
    BFarray my_data;
    BFcomplex set_data[3][3] = 
        {{{5,1},{0,0},{100,0}},
        {{5,1},{30,0},{100,0}},
        {{30,0},{0,0},{10,1}}};
    BFcomplex** some_data;
    hipMalloc((void**)&some_data, sizeof(BFcomplex)*9);
    hipMemcpy(
        some_data, set_data, 
        sizeof(BFcomplex)*9, hipMemcpyHostToDevice);
    my_data.data = some_data;
    my_data.space = BF_SPACE_CUDA;
    my_data.shape[0] = 3;
    my_data.shape[1] = 3;
    my_data.dtype = 1;
    my_data.ndim = 2;
    my_data.strides[0] = 3*sizeof(BFcomplex);
    my_data.strides[1] = sizeof(BFcomplex);
    if (bfFFT(&my_data, &my_data, FFT_FORWARD) != BF_STATUS_SUCCESS)
    {
        printf("bfFFT failed!\n");
        return; 
    }
    hipfftComplex localdata[3][3]={};
    hipMemcpy(
        localdata, (hipfftComplex**)my_data.data, 
        sizeof(hipfftComplex)*9, hipMemcpyDeviceToHost);
    for(int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 3; j++)
                printf("%f\n",hipCrealf(localdata[i][j]));
    }
    //print successfully fft'd data.
    return;
}

void test_bffft_inverse_1d()
{
    BFarray my_data;
    BFcomplex set_data[5] = {{0,0},{30,0},{100,0},{30,0},{-5,0}};
    BFcomplex* some_data;
    hipMalloc((void**)&some_data, sizeof(BFcomplex)*5);
    hipMemcpy(some_data, set_data, sizeof(BFcomplex)*5, hipMemcpyHostToDevice);
    my_data.data = some_data;
    my_data.space = BF_SPACE_CUDA;
    my_data.shape[0] = 5;
    my_data.dtype = 1;
    my_data.ndim = 1;
    my_data.strides[0] = sizeof(BFcomplex);
    bfFFT(&my_data, &my_data, FFT_INVERSE);
    hipfftComplex localdata[5]={};
    hipMemcpy(localdata, (hipfftComplex*)my_data.data, sizeof(hipfftComplex)*5, hipMemcpyDeviceToHost);
    for(int i = 0; i < 5; i++)
        printf("%f+I%f\n",hipCrealf(localdata[i]),hipCimagf(localdata[i]));
    //print successfully fft'd data.
}

void test_bffft_1d()
{
    BFarray my_data;
    BFcomplex set_data[5] = {{0,0},{30,0},{100,0},{30,0},{-5,0}};
    BFcomplex* some_data;
    hipMalloc((void**)&some_data, sizeof(BFcomplex)*5);
    hipMemcpy(some_data, set_data, sizeof(BFcomplex)*5, hipMemcpyHostToDevice);
    my_data.data = some_data;
    my_data.space = BF_SPACE_CUDA;
    my_data.shape[0] = 5;
    my_data.dtype = 1;
    my_data.ndim = 1;
    my_data.strides[0] = sizeof(BFcomplex);
    bfFFT(&my_data, &my_data, FFT_FORWARD);
    hipfftComplex localdata[5]={};
    hipMemcpy(localdata, (hipfftComplex*)my_data.data, sizeof(hipfftComplex)*5, hipMemcpyDeviceToHost);
    for(int i = 0; i < 5; i++)
        printf("%f+I%f\n",hipCrealf(localdata[i]),hipCimagf(localdata[i]));
    //print successfully fft'd data.
}


int main()
{
    printf("Running...\n");
    //test_bffft_1d();
    //test_bffft_2d();
    //test_bffft_real();
    //test_bffft_inverse_1d();
    test_bffft_real_2d();
    printf("Done\n");
    return 0;
}
