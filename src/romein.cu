#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, The Bifrost Authors. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * * Redistributions of source code must retain the above copyright
 *   notice, this list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimer in the
 *   documentation and/or other materials provided with the distribution.
 * * Neither the name of The Bifrost Authors nor the names of its
 *   contributors may be used to endorse or promote products derived
 *   from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* 

Implements the Romein convolutional algorithm onto a GPU using CUDA. 

*/
#include <iostream>
#include <bifrost/romein.h>
#include "romein_kernels.cuh"

#include "assert.hpp"
#include "trace.hpp"
#include "utils.hpp"
#include "cuda.hpp"
#include "cuda/stream.hpp"

#include "Complex.hpp"

template<typename RealType>
__host__ __device__
inline Complex<RealType> Complexfcma(Complex<float> x, Complex<RealType> y, Complex<RealType> d) {
    RealType real_res;
    RealType imag_res;
    
    real_res = (x.x *  y.x) + d.x;
    imag_res = (x.x *  y.y) + d.y;
            
    real_res =  (x.y * y.y) + real_res;  
    imag_res = -(x.y * y.x) + imag_res;          
     
    return Complex<RealType>(real_res, imag_res);
}

template<typename InType, typename OutType>
__global__ void romein_kernel(int                           nbaseline,
                              int                           maxsupport, 
                              int                           gridsize, 
                              int                           nbatch,
                              const int3* __restrict__       positions,
                              const Complex32* __restrict__ kernels,
                              const InType* __restrict__    d_in,
                              OutType*                      d_out) {
    int batch_no = blockIdx.x;
    for(int i = threadIdx.x; i < maxsupport * maxsupport; i += blockDim.x) {
        int myU = i % maxsupport;
        int myV = i / maxsupport;
        
        int grid_point_u = myU;
        int grid_point_v = myV;
        OutType sum = OutType(0.0, 0.0);
        int vi_s = batch_no*nbaseline;
        int grid_s = batch_no*gridsize*gridsize;
        int vi = 0;
        for(vi = vi_s; vi < (vi_s+nbaseline); ++vi) {
            int3 uvw = positions[vi]; 
            
            // Determine convolution point. This is basically just an
            // optimised way to calculate.
            //int myConvU = myU - u;
            //int myConvV = myV - v;
            int myConvU = 0;
            int myConvV = 0;
            if( maxsupport > 1 ) {
                myConvU = (uvw.x - myU) % maxsupport;
                myConvV = (uvw.y - myV) % maxsupport;    
                if (myConvU < 0) myConvU += maxsupport;
                if (myConvV < 0) myConvV += maxsupport;
            } 
            
            // Determine grid point. Because of the above we know here that
            //   myGridU % max_supp = myU
            //   myGridV % max_supp = myV
            int myGridU = uvw.x + myConvU;
            int myGridV = uvw.y + myConvV;
            
            // Grid point changed?
            if (myGridU == grid_point_u && myGridV == grid_point_v) {
                // Nothin'
            } else {
                // Atomically add to grid. This is the bottleneck of this kernel.
                if( grid_point_u >= 0 && grid_point_u < gridsize && \
                    grid_point_v >= 0 && grid_point_v < gridsize ) {
                    atomicAdd(&d_out[grid_s + gridsize*grid_point_v + grid_point_u].x, sum.x);
                    atomicAdd(&d_out[grid_s + gridsize*grid_point_v + grid_point_u].y, sum.y);
                }
                // Switch to new point
                sum = OutType(0.0, 0.0);
                grid_point_u = myGridU;
                grid_point_v = myGridV;
            }
            
            //TODO: Re-do the w-kernel/gcf for our data.
            Complex32 px = kernels[vi*maxsupport*maxsupport + myConvV * maxsupport + myConvU];// ??
            // Sum up
            InType temp = d_in[vi];
            OutType vi_v = OutType(temp.x, temp.y);
            sum = Complexfcma(px, vi_v, sum);
        }
        
        if( grid_point_u >= 0 && grid_point_u < gridsize && \
            grid_point_v >= 0 && grid_point_v < gridsize ) {
            atomicAdd(&d_out[grid_s + gridsize*grid_point_v + grid_point_u].x, sum.x);
            atomicAdd(&d_out[grid_s + gridsize*grid_point_v + grid_point_u].y, sum.y);
        }
    }
}

template<typename InType, typename OutType>
inline void launch_romein_kernel(int        nbaseline,
                                 int        maxsupport, 
                                 int        gridsize, 
                                 int        nbatch,
                                 int*       positions,
                                 Complex32* kernels,
                                 InType*    d_in,
                                 OutType*   d_out,
                                 hipStream_t stream=0) {
    //cout << "LAUNCH for " << nelement << endl;
    // TODO: Is this really the best setup to use?
    dim3 block(8,1);
    dim3 grid(nbatch,1);
    /*
    cout << "  Block size is " << block.x << " by " << block.y << endl;
    cout << "  Grid  size is " << grid.x << " by " << grid.y << endl;
    */
    
    void* args[] = {&nbaseline,
                    &maxsupport,
                    &gridsize, 
                    &nbatch,
                    &positions,
                    &kernels,
                    &d_in,
                    &d_out};
    BF_CHECK_CUDA_EXCEPTION(hipLaunchKernel((void*)romein_kernel<InType,OutType>,
                                             grid, block,
                                             &args[0], 0, stream),
                            BF_STATUS_INTERNAL_ERROR);
}

class BFromein_impl {
    typedef int    IType;
    typedef double FType;
public: // HACK WAR for what looks like a bug in the CUDA 7.0 compiler
    typedef float  DType;
private:
    IType        _ntime;
    IType        _nchan;
    IType        _npol;
    IType        _nbaseline;
    IType        _maxsupport;
    IType        _gridsize;
    int*         _xyz = NULL;
    Complex32*   _kernels = NULL;
    hipStream_t _stream;
public:
    BFromein_impl() : _ntime(1), _nchan(1), _npol(1), _nbaseline(1), \
                      _maxsupport(1), _stream(g_cuda_stream) {}
    inline IType ntime()      const { return _ntime;      }
    inline IType nchan()      const { return _nchan;      }
    inline IType npol()       const { return _npol;       }
    inline IType nbaseline()  const { return _nbaseline;  }
    inline IType maxsupport() const { return _maxsupport; }
    inline IType gridsize()   const { return _gridsize;   }
    void init(IType ntime, 
              IType nchan, 
              IType npol,
              IType nbaseline,
              IType maxsupport, 
              IType gridsize) {
        BF_TRACE();
        _ntime      = ntime;
        _nchan      = nchan;
        _npol       = npol;
        _nbaseline  = nbaseline;
        _maxsupport = maxsupport;
        _gridsize   = gridsize;
    }
    void set_positions(BFarray const* positions) { 
        BF_TRACE();
        BF_TRACE_STREAM(_stream);
        BF_ASSERT_EXCEPTION(positions->dtype == BF_DTYPE_I32, BF_STATUS_UNSUPPORTED_DTYPE);
        
        _xyz = (int*) positions->data;
    }
    void set_kernels(BFarray const* kernels) {
        BF_TRACE();
        BF_TRACE_STREAM(_stream);
        BF_ASSERT_EXCEPTION(kernels->dtype == BF_DTYPE_CF32, BF_STATUS_UNSUPPORTED_DTYPE);
        
        _kernels = (Complex32*) kernels->data;
    }
    void execute(BFarray const* in, BFarray const* out) {
        BF_TRACE();
        BF_TRACE_STREAM(_stream);
        BF_ASSERT_EXCEPTION(_xyz != NULL, BF_STATUS_INVALID_STATE);
        BF_ASSERT_EXCEPTION(_kernels != NULL, BF_STATUS_INVALID_STATE);
        BF_ASSERT_EXCEPTION(out->dtype == BF_DTYPE_CF32 \
                                          || BF_DTYPE_CF64, BF_STATUS_UNSUPPORTED_DTYPE);
        
        BF_CHECK_CUDA_EXCEPTION(hipGetLastError(), BF_STATUS_INTERNAL_ERROR);
        
        int nbatch = in->shape[0]*in->shape[1]*in->shape[2];
        
#define LAUNCH_ROMEIN_KERNEL(IterType,OterType) \
        launch_romein_kernel(_nbaseline, _maxsupport, _gridsize, nbatch, \
                             _xyz, _kernels, \
                             (IterType)in->data, (OterType)out->data, \
                             _stream)
        
        switch( in->dtype ) {
            case BF_DTYPE_CI8:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_ROMEIN_KERNEL(char2*, Complex32*);  break;
                    case BF_DTYPE_CF64: LAUNCH_ROMEIN_KERNEL(char2*, Complex64*);  break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                };
                break;
            case BF_DTYPE_CI16:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_ROMEIN_KERNEL(short2*, Complex32*); break;
                    case BF_DTYPE_CF64: LAUNCH_ROMEIN_KERNEL(short2*, Complex64*); break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            case BF_DTYPE_CI32:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_ROMEIN_KERNEL(int2*, Complex32*); break;
                    case BF_DTYPE_CF64: LAUNCH_ROMEIN_KERNEL(int2*, Complex64*); break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            case BF_DTYPE_CI64:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_ROMEIN_KERNEL(long2*, Complex32*); break;
                    case BF_DTYPE_CF64: LAUNCH_ROMEIN_KERNEL(long2*, Complex64*); break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            case BF_DTYPE_CF32:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_ROMEIN_KERNEL(float2*, Complex32*);   break;
                    case BF_DTYPE_CF64: LAUNCH_ROMEIN_KERNEL(float2*, Complex64*);   break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            case BF_DTYPE_CF64:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_ROMEIN_KERNEL(double2*, Complex32*);  break;
                    case BF_DTYPE_CF64: LAUNCH_ROMEIN_KERNEL(double2*, Complex64*);  break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
        }
#undef LAUNCH_ROMEIN_KERNEL
        
        BF_CHECK_CUDA_EXCEPTION(hipGetLastError(), BF_STATUS_INTERNAL_ERROR);
    }
    void set_stream(hipStream_t stream) {
        _stream = stream;
    }
};

BFstatus bfRomeinCreate(BFromein* plan_ptr) {
    BF_TRACE();
    BF_ASSERT(plan_ptr, BF_STATUS_INVALID_POINTER);
    BF_TRY_RETURN_ELSE(*plan_ptr = new BFromein_impl(),
                       *plan_ptr = 0);
}

BFstatus bfRomeinInit(BFromein       plan,
                      BFarray const* positions,
                      BFarray const* kernels,
                      BFsize         gridsize) {
    BF_TRACE();
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    BF_ASSERT(positions,                BF_STATUS_INVALID_POINTER);
    BF_ASSERT(positions->ndim == 5,     BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(positions->shape[4] == 3, BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(space_accessible_from(positions->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    BF_ASSERT(kernels,                                BF_STATUS_INVALID_POINTER);
    BF_ASSERT(kernels->ndim == 6,                     BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(kernels->shape[4] == kernels->shape[5], BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(space_accessible_from(kernels->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    
    // Discover the dimensions of the positions/kernels.
    int ntime, nchan, npol, nbaseline, maxsupport;
    ntime = positions->shape[0];
    nchan = positions->shape[1];
    npol = positions->shape[2];
    nbaseline = positions->shape[3];
    maxsupport = kernels->shape[5];
    
    BF_TRY(plan->init(ntime, nchan, npol, nbaseline, maxsupport, gridsize));
    BF_TRY(plan->set_positions(positions));
    BF_TRY_RETURN(plan->set_kernels(kernels));
}
BFstatus bfRomeinSetStream(BFromein    plan,
                           void const* stream) {
    BF_TRACE();
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    BF_ASSERT(stream, BF_STATUS_INVALID_POINTER);
    BF_TRY_RETURN(plan->set_stream(*(hipStream_t*)stream));
}
BFstatus bfRomeinSetPositions(BFromein       plan,
                              BFarray const* positions) {
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    BF_ASSERT(positions,            BF_STATUS_INVALID_POINTER);
    BF_ASSERT(positions->ndim == 5, BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(positions->shape[0] == plan->ntime(),     BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(positions->shape[1] == plan->nchan(),     BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(positions->shape[2] == plan->npol(),      BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(positions->shape[3] == plan->nbaseline(), BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(positions->shape[4] == 3,                 BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(space_accessible_from(positions->space,   BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    BF_TRY_RETURN(plan->set_positions(positions));
}
BFstatus bfRomeinSetKernels(BFromein       plan, 
                            BFarray const* kernels) {
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    BF_ASSERT(kernels,            BF_STATUS_INVALID_POINTER);
    BF_ASSERT(kernels->ndim == 6, BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(kernels->shape[0] == plan->ntime(),      BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(kernels->shape[1] == plan->nchan(),      BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(kernels->shape[2] == plan->npol(),       BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(kernels->shape[3] == plan->nbaseline(),  BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(kernels->shape[4] == plan->maxsupport(), BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(kernels->shape[5] == plan->maxsupport(), BF_STATUS_INVALID_SHAPE  );
    
    BF_ASSERT(space_accessible_from(kernels->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    BF_TRY_RETURN(plan->set_kernels(kernels));
}
BFstatus bfRomeinExecute(BFromein          plan,
                         BFarray const* in,
                         BFarray const* out) {
    BF_TRACE();
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    BF_ASSERT(in,   BF_STATUS_INVALID_POINTER);
    BF_ASSERT(out,  BF_STATUS_INVALID_POINTER);
    BF_ASSERT( in->ndim == 4,          BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(out->ndim == in->ndim+1, BF_STATUS_INVALID_SHAPE);
    
//     TODO: What all should we support here?
//     BFarray in_flattened;
//     if( in->ndim > 4 ) {
//         // Keep the last three dim but attempt to flatten all others
//         unsigned long keep_dims_mask = padded_dims_mask(in);
//         keep_dims_mask |= 0x1 << (out->ndim-1);
//         keep_dims_mask |= 0x1 << (out->ndim-2);
//         keep_dims_mask |= 0x1 << (out->ndim-3);
//         keep_dims_mask |= 0x1 << (out->ndim-4);
//         flatten(in,   &in_flattened, keep_dims_mask);
//         in  =  &in_flattened;
//         BF_ASSERT(in_flattened.ndim == 4, BF_STATUS_UNSUPPORTED_SHAPE);
//     }
    BF_ASSERT( in->shape[0] == plan->ntime(),     BF_STATUS_INVALID_SHAPE);
    BF_ASSERT( in->shape[1] == plan->nchan(),     BF_STATUS_INVALID_SHAPE);
    BF_ASSERT( in->shape[2] == plan->npol(),      BF_STATUS_INVALID_SHAPE);
    BF_ASSERT( in->shape[3] == plan->nbaseline(), BF_STATUS_INVALID_SHAPE);
    
//     TODO: What all should we support here?
//     BFarray out_flattened;
//     if( out->ndim > 5 ) {
//         // Keep the last four dim but attempt to flatten all others
//         unsigned long keep_dims_mask = padded_dims_mask(out);
//         keep_dims_mask |= 0x1 << (out->ndim-1);
//         keep_dims_mask |= 0x1 << (out->ndim-2);
//         keep_dims_mask |= 0x1 << (out->ndim-3);
//         keep_dims_mask |= 0x1 << (out->ndim-4);
//         keep_dims_mask |= 0x1 << (out->ndim-5);
//         flatten(out,   &out_flattened, keep_dims_mask);
//         out  =  &out_flattened;
//         BF_ASSERT(out_flattened.ndim == 5, BF_STATUS_UNSUPPORTED_SHAPE);
//     }
    BF_ASSERT(out->shape[0] == in->shape[0],     BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(out->shape[1] == plan->nchan(),    BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(out->shape[2] == plan->npol(),     BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(out->shape[3] == plan->gridsize(), BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(out->shape[4] == plan->gridsize(), BF_STATUS_INVALID_SHAPE);
    
    BF_ASSERT(space_accessible_from( in->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    BF_ASSERT(space_accessible_from(out->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    BF_TRY_RETURN(plan->execute(in, out));
}

BFstatus bfRomeinDestroy(BFromein plan) {
    BF_TRACE();
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    delete plan;
    return BF_STATUS_SUCCESS;
}




// Fix this to use templates properly.
BFstatus romein_float(BFarray const* data, // Our data, strided by d
		      BFarray const* uvgrid, // Our UV Grid to Convolve onto, strided by g
		      BFarray const* illum, // Our convolution kernel.
		      BFarray const* data_xloc,
		      BFarray const* data_yloc,
		      BFarray const* data_zloc,
		      int max_support,
		      int grid_size,
		      int data_size,
		      int nbatch){


    //TODO: I think remove these as the overhead is probably quite high...
    BF_TRACE();
    //BF_ASSERT(uvgrid && data && illum && data_xloc && data_yloc && data_zloc,
    //	      BF_STATUS_UNSUPPORTED_DTYPE);
    BF_ASSERT(uvgrid->dtype == BF_DTYPE_CF32, BF_STATUS_UNSUPPORTED_DTYPE);
    BF_ASSERT(data->dtype == BF_DTYPE_CF32, BF_STATUS_UNSUPPORTED_DTYPE);
    BF_ASSERT(illum->dtype == BF_DTYPE_CF32, BF_STATUS_UNSUPPORTED_DTYPE);
    BF_ASSERT(data_xloc->dtype == BF_DTYPE_I32, BF_STATUS_UNSUPPORTED_DTYPE);
    BF_ASSERT(data_yloc->dtype == BF_DTYPE_I32, BF_STATUS_UNSUPPORTED_DTYPE);
    BF_ASSERT(data_zloc->dtype == BF_DTYPE_I32, BF_STATUS_UNSUPPORTED_DTYPE);


    BF_ASSERT(space_accessible_from(data->space, BF_SPACE_CUDA), BF_STATUS_UNSUPPORTED_SPACE);
    BF_ASSERT(space_accessible_from(uvgrid->space, BF_SPACE_CUDA), BF_STATUS_UNSUPPORTED_SPACE);
    BF_ASSERT(space_accessible_from(illum->space, BF_SPACE_CUDA), BF_STATUS_UNSUPPORTED_SPACE);
    BF_ASSERT(space_accessible_from(data_xloc->space, BF_SPACE_CUDA), BF_STATUS_UNSUPPORTED_SPACE);
    BF_ASSERT(space_accessible_from(data_yloc->space, BF_SPACE_CUDA), BF_STATUS_UNSUPPORTED_SPACE);
    BF_ASSERT(space_accessible_from(data_zloc->space, BF_SPACE_CUDA), BF_STATUS_UNSUPPORTED_SPACE);
    
    void const* dptr = data->data;
    void const* uvgridptr = uvgrid->data;
    void const* illumptr = illum->data;
    void const* xloc = data_xloc->data;
    void const* yloc = data_yloc->data;
    void const* zloc = data_zloc->data;
    cuda::child_stream stream(g_cuda_stream);
    BF_TRACE_STREAM(stream);
    
    scatter_grid_kernel <<< nbatch, 8, 0, stream >>> ((hipComplex*)dptr,
						  (hipComplex*)uvgridptr,
						  (hipComplex*)illumptr,
						  (int*)xloc,
						  (int*)yloc,
						  (int*)zloc,
						  max_support,
						  grid_size,
						  data_size);
    
    //hipError_t err = hipGetLastError();
    //std::cout << "Error: " << hipGetErrorString(err) << "\n";
    return BF_STATUS_SUCCESS;
}
		      
